#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2018-2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <torch/extension.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <math.h>
#include <algorithm>
#include <stdlib.h>
#include "cpu/vision.h"


/*rle cuda kernels are cuda version of the corresponding cpu functions here 
https://github.com/cocodataset/cocoapi/blob/master/common/maskApi.c 
these are only a subset of rle kernels.*/

typedef unsigned int uint;
typedef unsigned long siz;
typedef unsigned char byte;

//6144 is based on minimum shared memory size per SM 
//across all pytorch-supported GPUs. Need to use blocking
//to avoid this restriction
const int BUFFER_SIZE=6144;
const int CNTS_SIZE=6144;

__global__ void create_poly_rel_idx_kernel(int* segmask_poly_idx, int* segmask_poly_rel_idx, int64_t* clamped_idxs, int num_anchors,
				           int* per_anchor_poly_idx, int* per_anchor_first_poly_idx,
				           int* per_anchor_poly_rel_idx)
{
    int anchor_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (anchor_idx < num_anchors)
    {
	int segmask_id = static_cast<int>(clamped_idxs[anchor_idx]);
	int segmask_first_poly_idx = segmask_poly_idx[segmask_id];
	int num_mask = segmask_poly_idx[segmask_id+1] - segmask_first_poly_idx;
	int poly_start_idx = per_anchor_poly_idx[anchor_idx];
	int poly_rel_start_idx = per_anchor_first_poly_idx[anchor_idx];
	for (int poly_id = 0;  poly_id < num_mask;  ++poly_id)
	    per_anchor_poly_rel_idx[poly_start_idx+poly_id] = poly_rel_start_idx + (segmask_poly_rel_idx[segmask_first_poly_idx+poly_id] - segmask_poly_rel_idx[segmask_first_poly_idx]);
    } else if (anchor_idx == num_anchors) {
        int poly_start_idx = per_anchor_poly_idx[anchor_idx];
        int poly_rel_start_idx = per_anchor_first_poly_idx[anchor_idx];
        per_anchor_poly_rel_idx[poly_start_idx] = poly_rel_start_idx;
    }
}

template<typename T_i, typename T_o>
__global__ void create_dense_poly_data_kernel(int* segmask_poly_idx, int* segmask_poly_rel_idx, int64_t* clamped_idxs,
		                              int* per_anchor_poly_idx, int* per_anchor_poly_rel_idx,
				              T_i* segmask_dense_poly_data, T_o* dense_poly_data)
{
    int anchor_idx = blockIdx.x;
    int segmask_id = static_cast<int>(clamped_idxs[anchor_idx]);
    int segmask_first_poly_idx = segmask_poly_idx[segmask_id];
    int segmask_first_poly_rel_idx = segmask_poly_rel_idx[segmask_first_poly_idx];
    T_i* src = segmask_dense_poly_data + segmask_first_poly_rel_idx;
    int poly_start_idx = per_anchor_poly_idx[anchor_idx];
    int poly_rel_start_idx = per_anchor_poly_rel_idx[poly_start_idx];
    T_o* dst = dense_poly_data + poly_rel_start_idx;
    int poly_end_idx = per_anchor_poly_idx[anchor_idx+1];
    int num_samples = per_anchor_poly_rel_idx[poly_end_idx] - poly_rel_start_idx;
    for (int i = threadIdx.x;  i < num_samples;  i+= blockDim.x) {
        dst[i] = static_cast<T_o>(src[i]);
    }
}

__device__ void d_merge_masks_cuda_kernel(
	byte *masks_in, float *masks_out, const int mask_size,
	int anchor_idx, int mask_start_idx, int num_of_masks_to_merge
	)
{
    int tid = threadIdx.x;
    int jump_block = blockDim.x;
    
    for(int j = tid; j < mask_size * mask_size; j += jump_block){
        int transposed_pixel = (j % mask_size) * mask_size + j / mask_size;
        byte pixel = 0;
        for(int k = 0; k < num_of_masks_to_merge; k++){
            if (masks_in[(mask_start_idx + k) * mask_size * mask_size + j] == 1) pixel = 1;
            if (pixel == 1) break;
        }
        masks_out[anchor_idx * mask_size * mask_size + transposed_pixel] = (float) pixel;       
    }
}

//merging masks happens on mask format, not RLE format.
__global__ void merge_masks_cuda_kernel(byte *masks_in, float *masks_out, const int mask_size,
                                        int *per_anchor_poly_idx){

    int anchor_idx = blockIdx.x;
    int mask_start_idx = per_anchor_poly_idx[anchor_idx];
    int num_of_masks_to_merge = per_anchor_poly_idx[anchor_idx + 1]-per_anchor_poly_idx[anchor_idx];

    d_merge_masks_cuda_kernel(masks_in, masks_out, mask_size, anchor_idx, mask_start_idx, num_of_masks_to_merge);
}

//merging masks happens on mask format, not RLE format.
__global__ void global_transforms_merge_masks_cuda_kernel(
	byte *masks_in,
	float *masks_out,
	const int mask_size,
	const int* target_index,
	const int* indexes,
	const int64_t* clamped_idxs,
        const float* weights
	)
{
    const int anchor_idx = blockIdx.x;
    if (weights[anchor_idx] > 0.0f)
    {
        const int header_size = indexes[0];
        const int index = *target_index;
        const int mask_offset = indexes[header_size+index];
        int mask_start_idx = 0;
        for (int i = 0;  i < anchor_idx;  ++i) {
            int j = static_cast<int>(clamped_idxs[i]);
            mask_start_idx += (indexes[mask_offset+j+1] - indexes[mask_offset+j]); // +num_polygons for mask clamped_idxs[i]
        }
        const int num_of_masks_to_merge = indexes[mask_offset+clamped_idxs[anchor_idx]+1] - indexes[mask_offset+clamped_idxs[anchor_idx]];

        d_merge_masks_cuda_kernel(masks_in, masks_out, mask_size, anchor_idx, mask_start_idx, num_of_masks_to_merge);
    }
}

/*cuda version of rleFrPoly function in this API:
https://github.com/cocodataset/cocoapi/blob/master/common/maskApi.c
Merged with crop_and_scale + decode_rle
*/

__device__ void d_rle_fr_poly(
	int anchor_idx,
	int poly_id,
	int k,
	const float *xy,
	int h, int w,
	int *cnts,
	int *x_in, int *y_in, int *u_in, int *v_in, int *a_in, int *b_in,
	int *num_of_cnts, byte* mask,
	const float4 *anchor_data
	)
{
    int tid = threadIdx.x;
    int block_jump = blockDim.x;
    int cnts_offset = poly_id * CNTS_SIZE;

    int *x = x_in + poly_id * BUFFER_SIZE;
    int *y = y_in + poly_id * BUFFER_SIZE;
    int *u = u_in + poly_id * BUFFER_SIZE;
    int *v = v_in + poly_id * BUFFER_SIZE;
    int *a = a_in + poly_id * BUFFER_SIZE;
    int *b = b_in + poly_id * BUFFER_SIZE;
    /* upsample and get discrete points densely aint entire boundary */
    int j, m = 0;
    float scale = 5.0f;
    float4 anchor_bb = anchor_data[anchor_idx];
    float ratio_w = (scale * (float)w) / fmaxf(anchor_bb.z - anchor_bb.x, 1.0f);
    float ratio_h = (scale * (float)h) / fmaxf(anchor_bb.w - anchor_bb.y, 1.0f);
    __shared__ int shbuf1[BUFFER_SIZE];
    __shared__ int shbuf2[BUFFER_SIZE];
    for(int j = tid; j < BUFFER_SIZE; j += block_jump) {
        shbuf1[j] = 0; 
        shbuf2[j] = 0;
    }
    for(int j = tid; j <= k; j += block_jump) {
	x[j] = (int)(ratio_w*((j < k ? xy[2*j  ] : xy[0]) - anchor_bb.x) + 0.5f);
	y[j] = (int)(ratio_h*((j < k ? xy[2*j+1] : xy[1]) - anchor_bb.y) + 0.5f);
    }
    __syncthreads();
        
    for(int j = tid; j < k; j += block_jump){
        int xs = x[j], xe = x[j + 1], ys = y[j], ye = y[j + 1], dx, dy, t, dist;
        int flip; 
        double s; 
        dx = abs(xe - xs); 
        dy = abs(ys - ye);
        flip = (dx >= dy && xs > xe) || (dx < dy && ys > ye);
        if (flip) {t = xs; xs = xe; xe = t; t = ys; ys = ye; ye = t;}
        s = dx >= dy ? (double) (ye - ys) / dx : (double) (xe - xs) / dy;
        dist = dx >= dy ? dx + 1 : dy + 1;
        shbuf1[j + 1] = dist; 
        shbuf2[j + 1] = dist;
    }
    __syncthreads();
    //block-wide exclusive prefix scan
    int switch_buf = 0;
    for (int offset = 1; offset <= k; offset *= 2){
        switch_buf = 1 - switch_buf;
        if (switch_buf == 0){
            for(int j = tid; j <= k; j += block_jump){
                if (j >= offset) shbuf2[j] = shbuf1[j] + shbuf1[j - offset];
                else shbuf2[j] = shbuf1[j];                
            }
        }
        else if (switch_buf == 1){
            for(int j = tid; j <= k; j += block_jump){
                if (j >= offset) shbuf1[j] = shbuf2[j] + shbuf2[j - offset];
                else shbuf1[j] = shbuf2[j];                
            }                  
        } 
        __syncthreads();
    }
      
    for (int j = tid; j < k; j += block_jump){
        int xs = x[j], xe = x[j + 1], ys = y[j], ye = y[j + 1], dx, dy, t, d, dist;
        int flip; 
        double s; 
        dx = __sad(xe, xs, 0); 
        dy = __sad(ys, ye, 0);
        flip = (dx >= dy && xs > xe) || (dx < dy && ys > ye);
        if (flip) {t = xs; xs = xe; xe = t; t = ys; ys = ye; ye = t;}
        s = dx >= dy ? (double) (ye - ys) / dx : (double) (xe - xs) / dy;
        m = switch_buf == 0 ? shbuf2[j] : shbuf1[j];
        if (dx >= dy) for (d = 0; d <= dx; d++) {
          /*the multiplication statement 's*t' causes nvcc to optimize with flush-to-zero=True for 
          double precision multiply, which we observe produces different results than CPU occasionally. 
          To force flush-to-zero=False, we use __dmul_rn intrinsics function */
          t = flip ? dx - d : d; 
          u[m] = t + xs; 
          v[m] = (int) (ys + __dmul_rn(s, t) + .5); 
          m++; 
        } 
        else for (d = 0; d <= dy; d++) {
          t = flip ? dy - d : d; 
          v[m] = t + ys; 
          u[m] = (int) (xs + __dmul_rn(s, t) + .5); 
          m++;
        }
    }    
    __syncthreads();
    m = switch_buf == 0 ? shbuf2[k] : shbuf1[k];
    int k2 = m;
    __syncthreads();
    double xd, yd;
    if (tid == 0) {
        shbuf1[tid] = 0; 
        shbuf2[tid] = 0;
    }     
    /* get points aint y-boundary and downsample */
    for (int j = tid; j < k2; j += block_jump){
        if (j > 0){
            if (u[j] != u[j - 1]){
                xd = (double) (u[j] < u[j-1] ? u[j] : u[j] - 1); 
                xd = (xd + .5) / scale - .5;   
                if (floor(xd) != xd || xd < 0 || xd > w - 1 ) {
                    shbuf1[j] = 0; 
                    shbuf2[j] = 0; 
                    continue;
                }
                yd = (double) (v[j] < v[j - 1] ? v[j] : v[j - 1]); yd = (yd + .5) / scale - .5;
                if (yd < 0) yd = 0; 
                else if (yd > h) yd = h; yd = ceil(yd);                
                shbuf1[j] = 1; 
                shbuf2[j] = 1;               
            } else {
                shbuf1[j] = 0; 
                shbuf2[j] = 0;
            }                    
        }    
    }
    __syncthreads(); 
    //exclusive prefix scan
    switch_buf = 0;
    for (int offset = 1; offset < k2; offset *= 2){
        switch_buf = 1 - switch_buf;
        if (switch_buf == 0){
            for (int j = tid; j < k2; j += block_jump){
                if (j >= offset) shbuf2[j] = shbuf1[j - offset] + shbuf1[j];
                else shbuf2[j] = shbuf1[j];                
            }
        }
        else if (switch_buf == 1){
            for (int j = tid; j < k2; j += block_jump){
                if (j >= offset) shbuf1[j] = shbuf2[j - offset] + shbuf2[j];
                else shbuf1[j] = shbuf2[j];                
            }                  
        } 
        __syncthreads();             
    }
  
    for (int j = tid; j < k2; j += block_jump){
        if (j > 0){
            if(u[j] != u[j - 1]){
                xd = (double) (u[j] < u[j - 1] ? u[j] : u[j] - 1); 
                xd = (xd + .5) / scale - .5;
                if (floor(xd) != xd || xd < 0 || xd > w - 1) {continue;}
                yd = (double) (v[j] < v[j - 1] ? v[j] : v[j - 1]); 
                yd = (yd + .5) / scale - .5;
                if (yd < 0) yd = 0; 
                else if (yd > h) yd = h; yd = ceil(yd);                
                m = switch_buf == 0 ? shbuf2[j - 1]:shbuf1[j - 1];
                x[m] = (int) xd; 
                y[m] = (int) yd; 
                m++;                
            }                   
        }    
    }
    __syncthreads(); 
    
    /* compute rle encoding given y-boundary points */
    m = switch_buf == 0 ? shbuf2[k2 - 1] : shbuf1[k2 - 1]; 
    int k3 = m;
    for (int j = tid; j <= k3; j += block_jump){
       if (j < k3) a[j] = (int) (x[j] * (int) (h) + y[j]);
       else a[j] = (int)(h * w);
    }
    k3++;
    __syncthreads();

    //run brick sort on a for k3+1 element
    //load k3+1 elements of a into shared memory
    for(int j = tid; j < k3; j += block_jump) shbuf1[j]=a[j];
    __syncthreads();
    int a_temp;
    for (int r = 0; r <= k3 / 2; r++){
        int evenCas = k3 / 2;
        int oddCas = (k3 - 1) / 2;
        //start with 0, need (k3+1)/2 CAS
        for (int j = tid; j < evenCas; j += block_jump){
            if (shbuf1[2 * j] > shbuf1[2 * j + 1]){
                a_temp = shbuf1[2 * j];
                shbuf1[2 * j]=shbuf1[2 * j + 1];
                shbuf1[2 * j + 1] = a_temp;
            }
        }
        __syncthreads();
        //start with 1
        for (int j = tid; j < oddCas; j += block_jump){
            if (shbuf1[2 * j + 1] > shbuf1[2 * j + 2]){
                a_temp=shbuf1[2 * j + 1];
                shbuf1[2 * j + 1] = shbuf1[2 * j + 2];
                shbuf1[2 * j + 2]=a_temp;
            }
        }
        __syncthreads();
    }

    for(int j = tid; j < k3; j += block_jump) {
        if(j>0) shbuf2[j] = shbuf1[j - 1];
        else shbuf2[j] = 0;
    }
     __syncthreads();
    for(int j = tid; j < k3; j += block_jump){
        shbuf1[j] -= shbuf2[j];
    }
    __syncthreads();
    int *cnts_buf = cnts + cnts_offset;
    if (tid == 0){
        j = m = 0;
        cnts_buf[m++] = shbuf1[j++];
        while (j < k3) if (shbuf1[j] > 0) cnts_buf[m++] = shbuf1[j++]; else {
            j++; if (j < k3) cnts_buf[m - 1] += shbuf1[j++]; }
        num_of_cnts[poly_id] = m;
    }
    __syncthreads();

    //
    // decode rle
    //

    m = num_of_cnts[poly_id];
    byte *mask_ptr = mask + poly_id * h * w;

    //initialize shbuf for scan. first element is 0 (exclusive scan)
    for (int i = tid; i < CNTS_SIZE; i += block_jump){
        shbuf1[i] = (i <= m & i > 0) ? cnts_buf[i - 1]:0;
        shbuf2[i] = (i <= m & i > 0) ? cnts_buf[i - 1]:0;
    }
    __syncthreads();
    
    //double buffering for scan
    switch_buf = 0;
    for (int offset = 1; offset <= m; offset *= 2){
        switch_buf = 1 - switch_buf;
        if(switch_buf == 0){
            for(int j = tid;j <= m;j += block_jump){
                if(j >= offset) shbuf2[j] = shbuf1[j]+shbuf1[j - offset];
                else shbuf2[j] = shbuf1[j];
            }
        }else if (switch_buf == 1){
            for(int j = tid;j <= m;j += block_jump){
                if(j >= offset) shbuf1[j] = shbuf2[j] + shbuf2[j - offset];
                else shbuf1[j] = shbuf2[j];
            }
        }
        __syncthreads();
    }
    int *scanned_buf = switch_buf == 0 ? shbuf2 : shbuf1;

    //find which bin pixel j falls into , which determines the pixel value
    //use binary search
    for(int j = tid; j < h * w; j += block_jump){
        int min_idx = 0;
        int max_idx = m;
        int mid_idx = m / 2;
        while(max_idx > min_idx){
            if(j > scanned_buf[mid_idx]) {
                min_idx = mid_idx+1;
                mid_idx = (min_idx + max_idx) / 2;
            }
            else if (j < scanned_buf[mid_idx]) {
                max_idx = mid_idx;
                mid_idx = (min_idx + max_idx) / 2;
            }
            else {
                mid_idx++;
                break;
            }
        }
        int k = mid_idx;
        byte pixel = k % 2 == 0 ? 1 : 0;
        mask_ptr[j] = pixel;
    }
}

__global__ void rle_fr_poly_cuda_kernel(
	const float *dense_coordinates, int *poly_rel_idx, int h, int w,
	int *cnts, int *x_in, int *y_in, int *u_in, int *v_in, int *a_in,
	int *b_in, int *num_of_cnts, byte* mask,
	int *per_anchor_poly_idx, int max_num_poly_per_anchor,
	float4 *anchor_data
	)
{
    int anchor_idx = blockIdx.x / max_num_poly_per_anchor;
    int anchor_poly_id = blockIdx.x - anchor_idx * max_num_poly_per_anchor;
    int num_polys_for_this_anchor = per_anchor_poly_idx[anchor_idx+1] - per_anchor_poly_idx[anchor_idx];
    if (anchor_poly_id >= num_polys_for_this_anchor)
	    return;  // no work for this block
    int poly_id = per_anchor_poly_idx[anchor_idx] + anchor_poly_id;
    int k = (poly_rel_idx[poly_id + 1] - poly_rel_idx[poly_id]) / 2;
    const float *xy = dense_coordinates + poly_rel_idx[poly_id];

    d_rle_fr_poly(anchor_idx, poly_id, k, xy, h, w, cnts, x_in, y_in, u_in, v_in, a_in, b_in, num_of_cnts, mask, anchor_data);
}


__global__ void global_transforms_generate_mask_targets_cuda_kernel(
		const int* target_index, // index argument passed by hybrid data loader in get_target call.
		const int* indexes, // self.indexes from COCODatasetPyt
		const float *transformed_dense_coordinates, // self.after_transforms_dense_xy
		//const float *bboxes_and_labels, // self.after_transforms_bboxes_and_labels
		const int64_t* clamped_idxs,
                const float* weights,
		int h, int w,
		int *cnts, int *x_in, int *y_in, int *u_in, int *v_in, int *a_in,
		int *b_in, int *num_of_cnts, byte* mask,
		int max_num_poly_per_anchor,
		const float4 *anchor_data
		)
{
    const int anchor_idx = blockIdx.x / max_num_poly_per_anchor;
    if (weights[anchor_idx] > 0.0f)
    {
        const int anchor_poly_id = blockIdx.x - anchor_idx * max_num_poly_per_anchor;
        const int header_size = indexes[0];
        const int mask_offset = indexes[header_size+*target_index];
        const int mask_id = static_cast<int>(clamped_idxs[anchor_idx]);
        const int polygon_offset = indexes[mask_offset+mask_id];
        const int num_polygons = indexes[mask_offset+mask_id+1] - polygon_offset;
        if (anchor_poly_id >= num_polygons)
            return; // no work for this block

        // determine output polygon
        int poly_id = 0;
        for (int i = 0;  i < anchor_idx;  ++i) {
            int j = static_cast<int>(clamped_idxs[i]);
            poly_id += (indexes[mask_offset+j+1] - indexes[mask_offset+j]); // +num_polygons for mask clamped_idxs[i]
        }
        poly_id += anchor_poly_id;

        int sample_offset = indexes[polygon_offset+anchor_poly_id];
        int k = (indexes[polygon_offset+anchor_poly_id+1] - sample_offset) >> 1;
        const float *xy = transformed_dense_coordinates + sample_offset;

        d_rle_fr_poly(anchor_idx, poly_id, k, xy, h, w, cnts, x_in, y_in, u_in, v_in, a_in, b_in, num_of_cnts, mask, anchor_data);
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

at::Tensor global_transforms_generate_mask_targets_cuda(
	at::Tensor target_index,
	at::Tensor transformed_img_infos,
	at::Tensor indexes,
	at::Tensor transformed_dense_coordinates,
	at::Tensor clamped_idxs,
        at::Tensor weights,
	at::Tensor anchors,
	const int max_num_poly_per_anchor,
	const int mask_size
	)
{
    const int M = mask_size;
    assert (M < 32); 
    //if M >=32, shared memory buffer size may not be
    //sufficient. Need to fix this by blocking
    const int num_of_anchors = anchors.size(0);
    const int max_num_of_poly = num_of_anchors * max_num_poly_per_anchor;
    at::Tensor d_x_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_y_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_u_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_v_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_a_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_b_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_mask_t = torch::empty({M * M * max_num_of_poly}, torch::CUDA(at::kByte));
    auto result =  torch::zeros({num_of_anchors, M, M}, torch::CUDA(at::kFloat));
    at::Tensor d_num_of_counts_t = torch::empty({max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_cnts_t = torch::empty({CNTS_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));

    auto stream = at::cuda::getCurrentCUDAStream();

    global_transforms_generate_mask_targets_cuda_kernel<<<max_num_of_poly, 512, 0, stream.stream()>>>(
	    target_index.data_ptr<int>(),
	    indexes.data_ptr<int>(),
	    transformed_dense_coordinates.data_ptr<float>(),
	    clamped_idxs.data_ptr<int64_t>(),
            weights.data_ptr<float>(),
	    M, M,
	    d_cnts_t.data_ptr<int>(),
	    d_x_t.data_ptr<int>(),
	    d_y_t.data_ptr<int>(),
	    d_u_t.data_ptr<int>(),
	    d_v_t.data_ptr<int>(),
	    d_a_t.data_ptr<int>(),
	    d_b_t.data_ptr<int>(),
	    d_num_of_counts_t.data_ptr<int>(),
	    d_mask_t.data_ptr<byte>(),
	    max_num_poly_per_anchor,
	    (const float4*)(anchors.data_ptr<float>())
	    );

    // cannot be merged with above kernel because all blocks must finish before this kernel can launch.
    global_transforms_merge_masks_cuda_kernel<<<num_of_anchors, 256, 0, stream.stream()>>>(
	    d_mask_t.data<byte>(), 
	    result.data_ptr<float>(),
            M, 
	    target_index.data_ptr<int>(),
	    indexes.data_ptr<int>(),
	    clamped_idxs.data_ptr<int64_t>(),
            weights.data_ptr<float>());

    return result;
}

// TODO: Add launch codes. Need to determine max num polys for all samples.
// Can do this in COCODatasetPYT init routine.

at::Tensor generate_mask_targets_cuda(at::Tensor dense_vector, const std::vector<std::vector<at::Tensor>> polygons, 
                                      const at::Tensor anchors, const int mask_size){    
    const int M = mask_size;
    assert (M < 32); 
    //if M >=32, shared memory buffer size may not be
    //sufficient. Need to fix this by blocking    
    float *d_anchor_data = anchors.data_ptr<float>();
    int num_of_anchors = anchors.size(0);  
    auto options = torch::dtype(torch::kInt).device(torch::kCPU).pinned_memory(true);
    auto per_anchor_poly_idx = at::empty({num_of_anchors + 1}, options);
    int nn = 0, max_num_poly_per_anchor = 0;
    // NB!
    // This loop introduces a race condition if this function is called more than once per step.
    // Note that this function is called once per image. The code runs correctly because the original
    // code has a GPU-CPU sync per image, but will not work if this sync is removed. That's why
    // there is a syncfree_ version of this function.
    for (int i = 0; i < num_of_anchors; i++){
            *(per_anchor_poly_idx.data_ptr<int>() + i) = nn;
	    int this_num_poly = polygons[i].size();
	    if (this_num_poly > max_num_poly_per_anchor) max_num_poly_per_anchor = this_num_poly;
            nn += this_num_poly;
    }
    *(per_anchor_poly_idx.data_ptr<int>() + num_of_anchors) = nn;
    int max_num_of_poly = num_of_anchors * max_num_poly_per_anchor;

    auto poly_rel_idx = at::empty({max_num_of_poly + 1}, options);
    float *dense_poly_data = dense_vector.data_ptr<float>();
    int start_idx = 0;
    int poly_count = 0;
  
    for(int i = 0; i < polygons.size(); i++){
  	    for(int j=0; j < polygons[i].size(); j++) {
                    *(poly_rel_idx.data_ptr<int>() + poly_count) = start_idx;
  		    start_idx += polygons[i][j].size(0);
  		    poly_count++;
  	    }
    }    
    *(poly_rel_idx.data_ptr<int>() + poly_count) = start_idx;

    at::Tensor d_x_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_y_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_u_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_v_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_a_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));//used with uint* pointer
    at::Tensor d_b_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt)); //used with uint* pointer
    at::Tensor d_mask_t = torch::empty({M * M * max_num_of_poly}, torch::CUDA(at::kByte));
    auto result =  torch::empty({num_of_anchors, M, M}, torch::CUDA(at::kFloat));
    at::Tensor d_num_of_counts_t = torch::empty({max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_cnts_t = torch::empty({CNTS_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    auto d_dense_vector = dense_vector.cuda();
    auto d_per_anchor_poly_idx = per_anchor_poly_idx.to(torch::kCUDA, true);
    auto d_poly_rel_idx = poly_rel_idx.to(torch::kCUDA, true);
    auto stream = at::cuda::getCurrentCUDAStream();
    
    //TODO: larger threads-per-block might be better here, because each CTA uses 32 KB of shmem,
    //and occupancy is likely shmem capacity bound                                                                                
    rle_fr_poly_cuda_kernel<<<max_num_of_poly, 512, 0, stream.stream()>>>(d_dense_vector.data_ptr<float>(),
                                                                   d_poly_rel_idx.data_ptr<int>(),
                                                                   M, M,
                                                                   d_cnts_t.data_ptr<int>(),
                                                                   d_x_t.data_ptr<int>(),
                                                                   d_y_t.data_ptr<int>(),
                                                                   d_u_t.data_ptr<int>(),
                                                                   d_v_t.data_ptr<int>(),
                                                                   d_a_t.data_ptr<int>(),
                                                                   d_b_t.data_ptr<int>(),
                                                                   d_num_of_counts_t.data_ptr<int>(),
								   d_mask_t.data_ptr<byte>(),
								   d_per_anchor_poly_idx.data_ptr<int>(),
								   max_num_poly_per_anchor,
								   (float4*) d_anchor_data);
                                                                 
    merge_masks_cuda_kernel<<<num_of_anchors, 256, 0, stream.stream()>>>(d_mask_t.data<byte>(), result.data_ptr<float>(),
                                                                      M, d_per_anchor_poly_idx.data_ptr<int>());
    return result;
}

at::Tensor syncfree_generate_mask_targets_cuda(at::Tensor clamped_idxs, const std::vector<std::vector<at::Tensor>> polygons, const at::Tensor anchors, const int mask_size)
{
    const int M = mask_size;
    assert (M < 32);
    //if M >=32, shared memory buffer size may not be
    //sufficient. Need to fix this by blocking

    // dimensions
    int num_masks = polygons.size();
    int num_of_anchors = anchors.size(0);
    int max_num_poly_per_anchor = 0, max_samples_per_anchor = 0, segmask_num_poly = 0;
    for (int i = 0;  i < polygons.size();  ++i) {
        int num_of_poly = polygons[i].size();
        segmask_num_poly += num_of_poly;
        max_num_poly_per_anchor = num_of_poly > max_num_poly_per_anchor ? num_of_poly : max_num_poly_per_anchor;
        int num_samples_this_anchor = 0;
        for (int j = 0;  j < polygons[i].size();  ++j) {
            int num_samples_this_poly = polygons[i][j].size(0);
            num_samples_this_anchor += num_samples_this_poly;
        }
        max_samples_per_anchor = num_samples_this_anchor > max_samples_per_anchor ? num_samples_this_anchor : max_samples_per_anchor;
    }
    int max_num_of_poly = num_of_anchors * max_num_poly_per_anchor;
    assert(num_of_anchors == clamped_idxs.numel());
    if (num_masks <= 0 || num_of_anchors <= 0 || max_num_poly_per_anchor <= 0 || max_samples_per_anchor <= 0 || segmask_num_poly <= 0) {
        printf("num_masks=%d, num_of_anchors=%d, max_num_poly_per_anchor=%d, max_samples_per_anchor=%d, segmask_num_poly=%d\n",num_masks,num_of_anchors,max_num_poly_per_anchor,max_samples_per_anchor,segmask_num_poly);
    }

    // create input tensors for ROI kernels.
    std::vector<at::Tensor> poly_vec;
    std::vector<int> polygons_per_segmask(num_masks, 0);
    std::vector<int> samples_per_segmask_polygon(segmask_num_poly+1, 0);
    std::vector<int> samples_per_segmask(num_masks, 0);
    for (int i = 0, k = 1, acc = 0;  i < num_masks;  ++i) {
	polygons_per_segmask[i] = polygons[i].size();
        int num_samples_this_segmask = 0;
	for (int j = 0;  j < polygons[i].size();  ++j) {
	    poly_vec.push_back(polygons[i][j]);
	    int num_samples_this_polygon = polygons[i][j].size(0);
	    num_samples_this_segmask += num_samples_this_polygon;
	    acc += num_samples_this_polygon;
	    samples_per_segmask_polygon[k++] = acc;
	}
	samples_per_segmask[i] = num_samples_this_segmask;
    }

    auto per_segmask_dense_poly_data = at::cat(poly_vec, 0);

    auto options = torch::dtype(torch::kInt).device(torch::kCPU).pinned_memory(true);
    auto per_segmask_poly_idx = at::tensor(polygons_per_segmask, options).to(torch::kCUDA, true);
    auto per_segmask_poly_rel_idx = at::tensor(samples_per_segmask_polygon, options).to(torch::kCUDA, true);
    auto per_segmask_num_samples = at::tensor(samples_per_segmask, options).to(torch::kCUDA, true);

    auto per_anchor_poly_idx = per_segmask_poly_idx.index_select(0, clamped_idxs);
    auto per_anchor_first_poly_rel_idx = per_segmask_num_samples.index_select(0, clamped_idxs);

    per_anchor_poly_idx = at::cumsum(per_anchor_poly_idx, 0, torch::kInt);
    per_anchor_poly_idx = at::cat({at::zeros({1},torch::CUDA(at::kInt)), per_anchor_poly_idx}, 0);

    per_segmask_poly_idx = at::cumsum(per_segmask_poly_idx, 0, torch::kInt);
    per_segmask_poly_idx = at::cat({at::zeros({1},torch::CUDA(at::kInt)), per_segmask_poly_idx}, 0);

    per_anchor_first_poly_rel_idx = at::cumsum(per_anchor_first_poly_rel_idx, 0, torch::kInt);
    per_anchor_first_poly_rel_idx = at::cat({at::zeros({1},torch::CUDA(at::kInt)), per_anchor_first_poly_rel_idx}, 0);

    // populate input tensors.
    auto stream = at::cuda::getCurrentCUDAStream();
    int num_blocks = (num_of_anchors + 255) / 256;
    auto per_anchor_poly_rel_idx = at::empty({max_num_of_poly + 1}, torch::CUDA(at::kInt));
    create_poly_rel_idx_kernel<<<num_blocks, 256, 0, stream.stream()>>>(
	    per_segmask_poly_idx.data_ptr<int>(),
	    per_segmask_poly_rel_idx.data_ptr<int>(),
	    clamped_idxs.data_ptr<int64_t>(),
	    num_of_anchors,
	    per_anchor_poly_idx.data_ptr<int>(),
	    per_anchor_first_poly_rel_idx.data_ptr<int>(),
	    per_anchor_poly_rel_idx.data_ptr<int>());
    auto per_anchor_dense_poly_data = at::empty({num_of_anchors * max_samples_per_anchor}, torch::CUDA(at::kFloat));
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(per_segmask_dense_poly_data.scalar_type(), "create_dense_poly_data_kernel", [&](){
            create_dense_poly_data_kernel<<<num_of_anchors, 256, 0, stream.stream()>>>(
	        per_segmask_poly_idx.data_ptr<int>(),
	        per_segmask_poly_rel_idx.data_ptr<int>(),
	        clamped_idxs.data_ptr<int64_t>(),
	        per_anchor_poly_idx.data_ptr<int>(),
	        per_anchor_poly_rel_idx.data_ptr<int>(),
                per_segmask_dense_poly_data.data_ptr<scalar_t>(),
                per_anchor_dense_poly_data.data_ptr<float>());
	    });

    // call kernels
    at::Tensor d_x_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_y_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_u_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_v_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_a_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));//used with uint* pointer
    at::Tensor d_b_t = torch::empty({BUFFER_SIZE * max_num_of_poly}, torch::CUDA(at::kInt)); //used with uint* pointer
    at::Tensor d_mask_t = torch::empty({M * M * max_num_of_poly}, torch::CUDA(at::kByte));
    auto result =  torch::empty({num_of_anchors, M, M}, torch::CUDA(at::kFloat));
    at::Tensor d_num_of_counts_t = torch::empty({max_num_of_poly}, torch::CUDA(at::kInt));
    at::Tensor d_cnts_t = torch::empty({CNTS_SIZE * max_num_of_poly}, torch::CUDA(at::kInt));

    float *d_anchor_data = anchors.data_ptr<float>();

    //TODO: larger threads-per-block might be better here, because each CTA uses 32 KB of shmem,
    //and occupancy is likely shmem capacity bound
    rle_fr_poly_cuda_kernel<<<max_num_of_poly, 512, 0, stream.stream()>>>(per_anchor_dense_poly_data.data_ptr<float>(),
                                                                   per_anchor_poly_rel_idx.data_ptr<int>(),
                                                                   M, M,
                                                                   d_cnts_t.data_ptr<int>(),
                                                                   d_x_t.data_ptr<int>(),
                                                                   d_y_t.data_ptr<int>(),
                                                                   d_u_t.data_ptr<int>(),
                                                                   d_v_t.data_ptr<int>(),
                                                                   d_a_t.data_ptr<int>(),
                                                                   d_b_t.data_ptr<int>(),
                                                                   d_num_of_counts_t.data_ptr<int>(),
								   d_mask_t.data_ptr<byte>(),
								   per_anchor_poly_idx.data_ptr<int>(),
								   max_num_poly_per_anchor,
								   (float4*) anchors.data_ptr<float>());

    merge_masks_cuda_kernel<<<num_of_anchors, 256, 0, stream.stream()>>>(d_mask_t.data<byte>(), result.data_ptr<float>(),
                                                                      M, per_anchor_poly_idx.data_ptr<int>());
    return result;
}
